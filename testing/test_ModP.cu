#include "hip/hip_runtime.h"
/*
The MIT License (MIT)

Copyright (c) 2015 Wei Dai

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

// Test all inline device mod P = 2^64-2^32+1 operations.

#include "../cuHE/ModP.h"
#include "../cuHE/Debug.h"
#include <time.h>
#include <stdio.h>
#include <NTL/ZZ.h>
NTL_CLIENT

#define num (1024*1024)
const ZZ P = to_ZZ(0xffffffff00000001);

void rand_array(uint64 *ptr) {
	for (int i=0; i<num; i++) {
		ptr[i] = rand();
		ptr[i] <<= 32*(rand()%2);
		ptr[i] |= rand();
	}
}
void rand_offset(int *l) {
	for (int i=0; i<num; i++) {
		l[i] = (rand()%8)*(rand()%8)*3;
	}
}
void rand_exp(int *e) {
	for (int i=0; i<num; i++) {
		e[i] = 1;//(unsigned)rand()>>1;
		if (e[i] < 0) {
			printf("Error: random exponent has opposite value.\n");
			exit(-1);
		}
	}
}
__global__ void _kernel_ls_modP(uint64 *dst, uint64 *src, int *offset) {
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < num) {
		uint64 t = cuHE::_ls_modP(src[idx], offset[idx]);
		dst[idx] = t;
	}
}
bool _test_ls_modP(uint64 *z, uint64 *x, int *l) {
	_kernel_ls_modP<<<(num+1023)/1024, 1024>>>(z, x, l);
	CCE();
	CSC(hipDeviceSynchronize());
	ZZ temp;
	for (int i=0; i<num; i++) {
		conv(temp, x[i]);
		temp <<= l[i];
		temp %= P;
		if (temp != to_ZZ(z[i])) {
			return false;
		}
	}
	return true;
}
__global__ void _kernel_add_modP(uint64 *dst, uint64 *src0, uint64 *src1) {
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < num) {
		uint64 t = cuHE::_add_modP(src0[idx], src1[idx]);
		dst[idx] = t;
	}
}
bool _test_add_modP(uint64 *z, uint64 *x, uint64 *y) {
	_kernel_add_modP<<<(num+1023)/1024, 1024>>>(z, x, y);
	CCE();
	CSC(hipDeviceSynchronize());
	ZZ temp;
	for (int i=0; i<num; i++) {
		conv(temp, x[i]);
		temp += y[i];
		temp %= P;
		if (temp != to_ZZ(z[i])) {
			return false;
		}
	}
	return true;
}
__global__ void _kernel_sub_modP(uint64 *dst, uint64 *src0, uint64 *src1) {
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < num) {
		uint64 t = cuHE::_sub_modP(src0[idx], src1[idx]);
		dst[idx] = t;
	}
}
bool _test_sub_modP(uint64 *z, uint64 *x, uint64 *y) {
	_kernel_sub_modP<<<(num+1023)/1024, 1024>>>(z, x, y);
	CCE();
	CSC(hipDeviceSynchronize());
	ZZ temp;
	for (int i=0; i<num; i++) {
		conv(temp, x[i]);
		temp -= y[i];
		temp %= P;
		if (temp != to_ZZ(z[i])) {
			return false;
		}
	}
	return true;
}
__global__ void _kernel_mul_modP(uint64 *dst, uint64 *src0, uint64 *src1) {
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < num) {
		uint64 t = cuHE::_mul_modP(src0[idx], src1[idx]);
		dst[idx] = t;
	}
}
bool _test_mul_modP(uint64 *z, uint64 *x, uint64 *y) {
	_kernel_mul_modP<<<(num+1023)/1024, 1024>>>(z, x, y);
	CCE();
	CSC(hipDeviceSynchronize());
	ZZ temp;
	for (int i=0; i<num; i++) {
		conv(temp, x[i]);
		temp *= y[i];
		temp %= P;
		if (temp != to_ZZ(z[i])) {
			return false;
		}
	}
	return true;
}
__global__ void _kernel_pow_modP(uint64 *dst, uint64 *src, int *exp) {
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < num) {
		uint64 t = cuHE::_pow_modP(src[idx], exp[idx]);
		dst[idx] = t;
	}
}
bool _test_pow_modP(uint64 *z, uint64 *x, int *e) {
	_kernel_pow_modP<<<(num+1023)/1024, 1024>>>(z, x, e);
	CCE();
	CSC(hipDeviceSynchronize());
	ZZ temp;
	for (int i=0; i<num; i++) {
		temp = PowerMod(to_ZZ(x[i]), e[i], P);
		if (temp != to_ZZ(z[i])) {
			return false;
		}
	}
	return true;
}
void print(bool result) {
	if (result)
		printf("pass\n");
	else
		printf("fail\n");
}
int main() {
	uint64 *x;
	uint64 *y;
	uint64 *z;
	int *l;
	int *e;
	CSC(hipMallocManaged(&x, num*sizeof(uint64)));
	CSC(hipMallocManaged(&y, num*sizeof(uint64)));
	CSC(hipMallocManaged(&z, num*sizeof(uint64)));
	CSC(hipMallocManaged(&l, num*sizeof(int)));
	CSC(hipMallocManaged(&e, num*sizeof(int)));
	srand(time(NULL));
	rand_array(x);
	rand_array(y);
	rand_offset(l);
	rand_exp(e);
	printf("_ls_modP:\t");
	print(_test_ls_modP(z, x, l));
	printf("_add_modP:\t");
	print(_test_add_modP(z, x, y));
	printf("_sub_modP:\t");
	print(_test_sub_modP(z, x, y));
	printf("_mul_modP:\t");
	print(_test_mul_modP(z, x, y));
	printf("_pow_modP:\t");
	print(_test_pow_modP(z, x, e));
	CSC(hipFree(x));
	CSC(hipFree(y));
	CSC(hipFree(z));
	CSC(hipFree(l));
	CSC(hipFree(e));
	return 0;
}
