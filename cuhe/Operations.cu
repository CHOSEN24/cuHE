/*
 *	The MIT License (MIT)
 *	Copyright (c) 2013-2015 Wei Dai
 *
 *	Permission is hereby granted, free of charge, to any person obtaining a copy
 *	of this software and associated documentation files (the "Software"), to deal
 *	in the Software without restriction, including without limitation the rights
 *	to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 *	copies of the Software, and to permit persons to whom the Software is
 *	furnished to do so, subject to the following conditions:
 *
 *	The above copyright notice and this permission notice shall be included in
 *	all copies or substantial portions of the Software.
 *
 *	THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 *	IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 *	FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 *	AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 *	LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 *	OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 *	THE SOFTWARE.
 */
#include "Operations.h"
#include "Parameters.h"
#include "DeviceManager.h"
#include "Debug.h"
#include "Base.h"
#include "CuHE.h"

namespace cuHE {

///////////////////////////////////////////////////////////////////////////////
//// Pre-computation //////////////////////////////////////////////////////////
static ZZ* crtPrime; // decreasing?
static ZZ* coeffModulus; // decreasing
void genCrtPrimes() {
	int pnum = numCrtPrime();
	crtPrime = new ZZ[pnum];
	unsigned* h_p = new unsigned[pnum];
	int logmid = logCoeffMin()-(pnum-depth())*logCrtPrime();
	// after cutting, fairly larger primes
	ZZ temp = to_ZZ(0x1<<logCrtPrime())-1;
	for (int i=0; i<=pnum-depth()-1; i++) {
		while (!ProbPrime(temp, 10))
			temp --;
		conv(h_p[i], temp);
		crtPrime[i] = temp;
		temp --;
	}

	// mid
	ZZ tmid;
	if (logmid != logCrtPrime())
		tmid = to_ZZ(0x1<<logmid)-1;
	else
		tmid = temp;
	while (!ProbPrime(tmid, 10))
		tmid --;
	conv(h_p[pnum-depth()], tmid);
	crtPrime[pnum-depth()] = tmid;

	// for cutting
	if (logCoeffCut() == logmid)
		temp = tmid-1;
	else if (logCoeffCut() == logCrtPrime())
		temp --;
	else
		temp = to_ZZ(0x1<<logCoeffCut())-1;
	for (int i=pnum-depth()+1; i<pnum; i++) {
		while (!ProbPrime(temp, 10) || temp%to_ZZ(modMsg()) != 1)
			temp --;
		conv(h_p[i], temp);
		crtPrime[i] = temp;
		temp --;
	}

	preload_crt_p(h_p, pnum);
	delete [] h_p;
};
void genCoeffModuli() {
	int d = depth();
	int pnum = numCrtPrime();
	coeffModulus = new ZZ[d];
	for (int i=0; i<d; i++) {
		coeffModulus[i] = 1;
		for (int j=0; j<pnum-i; j++)
			coeffModulus[i] *= crtPrime[j];
	}
}
void genCrtInvPrimes() {
	int pnum = numCrtPrime();
	uint32 *h_pinv = new uint32[pnum*(pnum-1)/2];
	ZZ temp;
	for (int i=1; i<pnum; i++)
		for (int j=0; j<i; j++)
			conv(h_pinv[i*(i-1)/2+j], InvMod(crtPrime[i]%crtPrime[j], crtPrime[j]));
	preload_crt_invp(h_pinv, pnum*(pnum-1)/2);
	delete [] h_pinv;
}
static int* icrtLevel; // one int for each device
static struct IcrtConst {
	uint32 *q;
	uint32 *qp;
	uint32 *qpinv;
} **icrtConst;
void genIcrtByLevel(int lvl) {
	int pnum = numCrtPrime(lvl);
	int words_q = wordsCoeff(lvl);
	int words_qp = wordsCoeff(lvl+1);
	for (int dev=0; dev<numDevices(); dev++) {
		CSC(hipSetDevice(dev));
		CSC(hipHostMalloc(&icrtConst[dev][lvl].q,
				words_q*sizeof(uint32)));
		CSC(hipHostMalloc(&icrtConst[dev][lvl].qp,
				pnum*words_qp*sizeof(uint32)));
		CSC(hipHostMalloc(&icrtConst[dev][lvl].qpinv,
				pnum*sizeof(uint32)));
	}
	ZZ *z_qp = new ZZ[pnum];
	for (int i=0; i<pnum; i++)
		z_qp[i] = coeffModulus[lvl]/crtPrime[i];
	for (int dev=0; dev<numDevices(); dev++) {
		BytesFromZZ((uint8 *)icrtConst[dev][lvl].q,
				coeffModulus[lvl], words_q*sizeof(uint32));
		for (int i=0; i<pnum; i++) {
			BytesFromZZ((uint8 *)(&icrtConst[dev][lvl].qp[words_qp*i]),
					z_qp[i], words_qp*sizeof(uint32));
			conv(icrtConst[dev][lvl].qpinv[i],
					InvMod(z_qp[i]%crtPrime[i], crtPrime[i]));
		}
	}
	delete [] z_qp;
};
void genIcrt() {
	icrtConst = new IcrtConst *[numDevices()];
	icrtLevel = new int[numDevices()];
	for (int dev=0; dev<numDevices(); dev++) {
		icrtConst[dev] = new IcrtConst[depth()];
		icrtLevel[dev] = -1;
	}
	for (int i=0; i<depth(); i++)
		genIcrtByLevel(i);
};
void loadIcrtConst(int lvl, int dev, hipStream_t st) {
	if (icrtLevel[dev] != lvl) {
		int pnum = numCrtPrime(lvl);
		int words_q = wordsCoeff(lvl);
		int words_qp = wordsCoeff(lvl+1);
		CSC(hipSetDevice(dev));
		load_icrt_M(icrtConst[dev][lvl].q, words_q, dev, st);
		load_icrt_mi(icrtConst[dev][lvl].qp, words_qp*pnum, dev, st);
		load_icrt_bi(icrtConst[dev][lvl].qpinv, pnum, dev, st);
		icrtLevel[dev] = lvl;
	}
};
void getCoeffModuli(ZZ* dst) {
	for (int i=0; i<depth(); i++)
		dst[i] = coeffModulus[i];
}
void initCrt(ZZ* coeffModulus) {
	genCrtPrimes();
	genCoeffModuli();
	genCrtInvPrimes();
	genIcrt();
	for (int dev=0; dev<numDevices(); dev++)
		loadIcrtConst(0, dev);
	getCoeffModuli(coeffModulus);
}
///////////////////////////////////////////////////////////////////////////////
static uint64 **d_swap; // conversion buffer
static uint32 **d_hold; // intt result buffer
void initNtt() {
	// twiddle factors
	const ZZ P = to_ZZ(0xffffffff00000001);
	const ZZ g = to_ZZ((uint64)15893793146607301539);
	int e0 = 65536/nttLen();
	ZZ w0 =	PowerMod(g, e0, P);
	uint64 *h_roots = new uint64[nttLen()];
	for (int i=0; i<nttLen(); i++)
		conv(h_roots[i], PowerMod(w0, i, P));
	preload_ntt(h_roots, nttLen());
	delete [] h_roots;
	// temporary result allocation
	d_swap = new uint64 *[numDevices()];
	d_hold = new uint32 *[numDevices()];
	for (int dev=0; dev<numDevices(); dev++) {
		hipSetDevice(dev);
		CSC(hipMalloc(&d_swap[dev], nttLen()*sizeof(uint64)));
		CSC(hipMalloc(&d_hold[dev], numCrtPrime()*nttLen()*sizeof(uint32)));
	}
}
uint32 *inttResult(int dev) {
	return ptrNttHold(dev);
}
uint64 **ptrNttSwap() { return d_swap;}
uint32 **ptrNttHold() {	return d_hold;}
uint64 *ptrNttSwap(int dev) { return d_swap[dev];}
uint32 *ptrNttHold(int dev) { return d_hold[dev];}
///////////////////////////////////////////////////////////////////////////////
uint64 **d_barrett_ntt;
uint32 **d_barrett_crt;
uint32 **d_barrett_src;
void createBarrettTemporySpace() {
	d_barrett_crt = new uint32*[numDevices()];
	d_barrett_ntt = new uint64*[numDevices()];
	d_barrett_src = new uint32*[numDevices()];
	for (int dev=0; dev<numDevices(); dev++) {
		hipSetDevice(dev);
		CSC(hipMalloc(&d_barrett_crt[dev], numCrtPrime()*nttLen()*sizeof(uint32)));
		CSC(hipMalloc(&d_barrett_ntt[dev], numCrtPrime()*nttLen()*sizeof(uint64)));
		CSC(hipMalloc(&d_barrett_src[dev], numCrtPrime()*nttLen()*sizeof(uint32)));
	}
}
static uint32 *ptrBarrettCrt(int dev) { return d_barrett_crt[dev];}
static uint64 *ptrBarrettNtt(int dev) { return d_barrett_ntt[dev];}
static uint32 *ptrBarrettSrc(int dev) { return d_barrett_src[dev];}
void setPolyModulus(ZZX m) {
	// compute NTL type zm, zu
	ZZ zq = coeffModulus[0];
	ZZX zm = m;
	ZZX zu;
	SetCoeff(zu, 2*modLen()-1, 1);
	zu /= zm;
	for (int i=0; i<=deg(zm); i++)
		SetCoeff(zm, i, coeff(zm, i)%zq);
	for (int i=0; i<=deg(zu); i++)
		SetCoeff(zu, i, coeff(zu, i)%zq);
	SetCoeff(zm, modLen(), 0);
	// prep m
	CuCtxt c;
	c.set(logCoeff(0), 0, zm);
	c.x2c();
	preload_barrett_m_c(c.cRep(), numCrtPrime()*crtLen()*sizeof(uint32));
	c.x2n();
	preload_barrett_m_n(c.nRep(), numCrtPrime()*nttLen()*sizeof(uint64));
	c.~CuCtxt();	
	// prep u
	CuCtxt cc;
	cc.set(logCoeff(0), 0, zu);
	cc.x2n();
	preload_barrett_u_n(cc.nRep(), numCrtPrime()*nttLen()*sizeof(uint64));
	cc.~CuCtxt();
};
void initBarrett(ZZX m) {
	setPolyModulus(m);
	createBarrettTemporySpace();
}

///////////////////////////////////////////////////////////////////////////////
//// Operations ///////////////////////////////////////////////////////////////
void crt(uint32 *dst, uint32 *src, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	hipSetDevice(dev);
	crt<<<(modLen()+63)/64, 64, wordsCoeff(lvl)*sizeof(uint32)*64, st>>>
			(dst, src, numCrtPrime(lvl), wordsCoeff(lvl), modLen(), crtLen());
	CCE();
}
void icrt(uint32 *dst, uint32 *src, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	loadIcrtConst(lvl, dev, st);
	CSC(hipStreamSynchronize(st));
	CSC(hipSetDevice(dev));
	icrt<<<(modLen()+63)/64, 64, 0, st>>>(dst, src, numCrtPrime(lvl),
			wordsCoeff(lvl), wordsCoeff(lvl+1), modLen(), crtLen());
	CCE();
}
void crtAdd(uint32 *sum, uint32 *x, uint32 *y, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	hipSetDevice(dev);
	crt_add<<<(modLen()+63)/64, 64, 0, st>>>(sum, x, y, numCrtPrime(lvl), modLen(), crtLen());
	CCE();
}
void crtAddInt(uint32 *sum, uint32 *x, unsigned a, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	hipSetDevice(dev);
	crt_add_int<<<(numCrtPrime(lvl)+63)/64, 64, 0, st>>>(sum, x, a, numCrtPrime(lvl), crtLen());
	CCE();
}
void crtAddNX1(uint32 *sum, uint32 *x, uint32 *scalar, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	hipSetDevice(dev);
	crt_add_nx1<<<(modLen()+63)/64, 64, 0, st>>>(sum, x, scalar, numCrtPrime(lvl), modLen(), crtLen());
	CCE();
}
void crtMulInt(uint32 *prod, uint32 *x, int a, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	hipSetDevice(dev);
	crt_mul_int<<<(numCrtPrime()-lvl+63)/64, 64, 0, st>>>(prod, x, a, numCrtPrime(lvl), crtLen());
	CCE();
}
void crtModSwitch(uint32 *dst, uint32 *src, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	hipSetDevice(dev);
	modswitch<<<(modLen()+63)/64, 64, 0, st>>>(dst, src, numCrtPrime(lvl),
			modLen(), crtLen(), modMsg());
	CCE();
}

//// single crt polynomial
void _ntt(uint64 *X, uint32 *x, int dev, hipStream_t st) {
	if (nttLen() == 16384) {
		ntt_1_16k_ext<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev), x);
		CCE();
		ntt_2_16k<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		ntt_3_16k<<<nttLen()/512, 64, 0, st>>>(X, ptrNttSwap(dev));
		CCE();
	}
	else if (nttLen() == 32768) {
		ntt_1_32k_ext<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev), x);
		CCE();
		ntt_2_32k<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		ntt_3_32k<<<nttLen()/512, 64, 0, st>>>(X, ptrNttSwap(dev));
		CCE();
	}
	else if (nttLen() == 65536) {
		ntt_1_64k_ext<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev), x);
		CCE();
		ntt_2_64k<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		ntt_3_64k<<<nttLen()/512, 64, 0, st>>>(X, ptrNttSwap(dev));
		CCE();
	}
}
void _nttw(uint64 *X, uint32 *x, int coeffwords, int relinIdx, int dev, hipStream_t st) {
	if (nttLen() == 16384) {
		ntt_1_16k_ext_block<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev), x, logRelin(), relinIdx, coeffwords);
		CCE();
		ntt_2_16k<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		ntt_3_16k<<<nttLen()/512, 64, 0, st>>>(X, ptrNttSwap(dev));
		CCE();
	}
	else if (nttLen() == 32768) {
		ntt_1_32k_ext_block<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev), x, logRelin(),relinIdx, coeffwords);
		CCE();
		ntt_2_32k<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		ntt_3_32k<<<nttLen()/512, 64, 0, st>>>(X, ptrNttSwap(dev));
		CCE();
	}
	else if (nttLen() == 65536) {
		ntt_1_64k_ext_block<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev), x, logRelin(),relinIdx, coeffwords);
		CCE();
		ntt_2_64k<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		ntt_3_64k<<<nttLen()/512, 64, 0, st>>>(X, ptrNttSwap(dev));
		CCE();
	}
}
// !!! x has length of nttLen()
void _intt(uint32 *x, uint64 *X, int crtidx, int dev, hipStream_t st) {
	if (nttLen() == 16384) {
		intt_1_16k<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev), X);
		CCE();
		ntt_2_16k<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		intt_3_16k_modcrt<<<nttLen()/512, 64, 0, st>>>(x, ptrNttSwap(dev), crtidx);
		CCE();
	}
	else if (nttLen() == 32768) {
		intt_1_32k<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev), X);
		CCE();
		ntt_2_32k<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		intt_3_32k_modcrt<<<nttLen()/512, 64, 0, st>>>(x, ptrNttSwap(dev), crtidx);
		CCE();
	}
	else if (nttLen() == 65536) {
		intt_1_64k<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev), X);
		CCE();
		ntt_2_64k<<<nttLen()/512, 64, 0, st>>>(ptrNttSwap(dev));
		CCE();
		intt_3_64k_modcrt<<<nttLen()/512, 64, 0, st>>>(x, ptrNttSwap(dev), crtidx);
		CCE();
	}
}
//// all crt polynomials
// ntt
void ntt(uint64 *X, uint32 *x, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	for (int i=0; i<numCrtPrime(lvl); i++)
		_ntt(X+i*nttLen(), x+i*crtLen(), dev, st);
}
void nttw(uint64 *X, uint32 *x, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	for (int i=0; i<numEvalKey(lvl); i++)
		_nttw(X+i*nttLen(), x, wordsCoeff(lvl), i, dev, st);
}
// intt holding result
void inttHold(uint64 *X, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	for (int i=0; i<numCrtPrime(lvl); i++)
		_intt(ptrNttHold(dev)+i*nttLen(), X+i*nttLen(), i, dev, st);
}
// intt without barrett copy result, x has nttLen()
void inttDoubleDeg(uint32 *x, uint64 *X, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	for (int i=0; i<numCrtPrime(lvl); i++)
		_intt(ptrNttHold(dev)+i*nttLen(), X+i*nttLen(), i, dev, st);
	CSC(hipMemcpyAsync(x, ptrNttHold(dev),
			numCrtPrime(lvl)*nttLen()*sizeof(uint32), hipMemcpyDeviceToDevice, st));
}
// intt without barrett copy result, x has crtLen()
void intt(uint32 *x, uint64 *X, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	for (int i=0; i<numCrtPrime(lvl); i++) {
		_intt(ptrNttHold(dev)+i*nttLen(), X+i*nttLen(), i, dev, st);
		CSC(hipMemcpyAsync(x+i*crtLen(), ptrNttHold(dev)+i*nttLen(),
			crtLen()*sizeof(uint32), hipMemcpyDeviceToDevice, st));
	}
}
// intt with barrett, x has crtLen()
void inttMod(uint32 *x, uint64 *X, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	for (int i=0; i<numCrtPrime(lvl); i++)
		_intt(ptrNttHold(dev)+i*nttLen(), X+i*nttLen(), i, dev, st);
	barrett(x, lvl, dev, st);
}
void nttMul(uint64 *z, uint64 *y, uint64 *x, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	ntt_mul<<<(nttLen()+63)/64, 64, 0, st>>>(z, y, x, numCrtPrime(lvl), nttLen());
}
void nttMulNX1(uint64 *z, uint64 *x, uint64 *scalar, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	ntt_mul_nx1<<<(nttLen()+63)/64, 64, 0, st>>>(z, x, scalar, numCrtPrime(lvl), nttLen());
}
void nttAdd(uint64 *z, uint64 *y, uint64 *x, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	ntt_add<<<(nttLen()+63)/64, 64, 0, st>>>(z, x, y, numCrtPrime(lvl), nttLen());
}
void nttAddNX1(uint64 *z, uint64 *x, uint64 *scalar, int logq, int dev, hipStream_t st) {
	int lvl = getLevel(logq);
	ntt_add_nx1<<<(nttLen()+63)/64, 64, 0, st>>>(z, x, scalar, numCrtPrime(lvl), nttLen());
}

void barrett(uint32 *dst, uint32 *src, int lvl, int dev, hipStream_t st) {
	hipSetDevice(dev);
	uint32 *ptrCrt = ptrBarrettCrt(dev);
	uint64 *ptrNtt = ptrBarrettNtt(dev);
	uint32 *ptrSrc = ptrBarrettSrc(dev);
	CSC(hipMemcpyAsync(ptrSrc, src, numCrtPrime(lvl)*nttLen()*sizeof(uint32),
				hipMemcpyDeviceToDevice, st));
	// ptrSrc = f, deg = 2n-2
	for (int i=0; i<numCrtPrime(lvl); i++)
		_ntt(ptrNtt+i*nttLen(), ptrSrc+i*nttLen()+modLen()-1, dev, st);
	// ptrNtt = f>>(n-1), deg = n-1
	barrett_mul_un<<<(nttLen()+63)/64, 64, 0, st>>>
			(ptrNtt, numCrtPrime(lvl), nttLen());
	inttDoubleDeg(ptrCrt, ptrNtt, logCoeff(lvl), dev, st);
	// ptrCrt = u * f>>(n-1), deg = 2n-2
	for (int i=0; i<numCrtPrime(lvl); i++)
		CSC(hipMemsetAsync(ptrCrt+i*nttLen(), 0, modLen()*sizeof(uint32), st));
	// ptrCrt = u*f>>(2n-1)<<n
	for (int i=0; i<numCrtPrime(lvl); i++)
		_ntt(ptrNtt+i*nttLen(), ptrCrt+i*nttLen()+modLen(), dev, st);
	// ptrNtt = (u * f>>(n-1))>>n = u*f>>(2n-1), deg = n-2
	barrett_mul_mn<<<(nttLen()+63)/64, 64, 0, st>>>
			(ptrNtt, numCrtPrime(lvl), nttLen());
	// ptrNtt = (m-x^n) * (u * f>>(n-1))>>n, deg = 2n-3
	barrett_sub_1<<<(modLen()+63)/64, 64, 0, st>>>
			(ptrSrc, ptrCrt, numCrtPrime(lvl), modLen(), nttLen());
	// ptrSrc = f - (u*f>>(2n-1))<<n
	inttDoubleDeg(ptrCrt, ptrNtt, logCoeff(lvl), dev, st);	
	// ptrCrt = (m-x^n) * (u * f>>(2n-1)), deg = 2n-3
	barrett_sub_2<<<(nttLen()+63)/64, 64, 0, st>>>
			(ptrSrc, ptrCrt, numCrtPrime(lvl), nttLen());
	// ptrSrc = f - (m*u*f)>>(2n-1), deg = n
	barrett_sub_mc<<<(nttLen()+63)/64, 64, numCrtPrime(lvl)*sizeof(uint32), st>>>
			(ptrSrc, numCrtPrime(lvl), modLen(), crtLen(), nttLen());
	// ptrSrc = ptrSrc - m, deg = n-1
	for (int i=0; i<numCrtPrime(lvl); i++)
		CSC(hipMemcpyAsync(dst+i*crtLen(), ptrSrc+i*nttLen(),
				crtLen()*sizeof(uint32), hipMemcpyDeviceToDevice, st));
}
void barrett(uint32 *dst, int lvl, int dev, hipStream_t st) {
	barrett(dst, inttResult(dev), lvl, dev, st);
}
} // end cuHE
